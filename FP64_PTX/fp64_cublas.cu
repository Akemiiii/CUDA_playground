#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

int main() {
    hipblasHandle_t handle;
    int m, n, k, i, j;
    double *h_A, *h_B, *h_C;
    double *d_A, *d_B, *d_C;
    double alpha = 1.0, beta = 0.0;

    // Specify the dimensions of the matrices
    m = 64;  // Number of rows of A and C
    n = 64;  // Number of columns of B and C
    k = 64;  // Number of columns of A and rows of B

    // Allocate host memory
    h_A = (double *)malloc(m * k * sizeof(double));
    h_B = (double *)malloc(k * n * sizeof(double));
    h_C = (double *)malloc(m * n * sizeof(double));

    // Initialize host matrices with sample values
    // For simplicity, matrices are filled with sequential numbers
    // A better approach is to fill them with random numbers or actual data
    for (i = 0; i < m; ++i) {
        for (j = 0; j < k; ++j) {
            h_A[i * k + j] = 1;//i * k + j + 1;
        }
    }

    for (i = 0; i < k; ++i) {
        for (j = 0; j < n; ++j) {
            h_B[i * n + j] = 1;//i * n + j + 1;
        }
    }

    // Initialize cuBLAS
    hipblasCreate(&handle);

    // Allocate device memory
    hipMalloc((void **)&d_A, m * k * sizeof(double));
    hipMalloc((void **)&d_B, k * n * sizeof(double));
    hipMalloc((void **)&d_C, m * n * sizeof(double));

    // Copy matrices from host to device
    hipblasSetMatrix(m, k, sizeof(double), h_A, m, d_A, m);
    hipblasSetMatrix(k, n, sizeof(double), h_B, k, d_B, k);

    // Perform matrix multiplication: C = alpha * A * B + beta * C
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_A, m, d_B, k, &beta, d_C, m);

    // Copy the result back to host
    hipblasGetMatrix(m, n, sizeof(double), d_C, m, h_C, m);

    // Print the result
    // printf("Result matrix C:\n");
    // for (i = 0; i < m; ++i) {
    //     for (j = 0; j < n; ++j) {
    //         printf("%f ", h_C[i * n + j]);
    //     }
    //     printf("\n");
    // }

    // Clean up
    free(h_A);
    free(h_B);
    free(h_C);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipblasDestroy(handle);

    return 0;
}
