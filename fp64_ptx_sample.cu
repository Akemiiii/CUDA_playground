#include<iostream>
#include<hip/hip_runtime.h>
#include<mma.h>
#include<assert.h>
#include"Common/hip/hip_runtime_api.h"

#define M 8
#define N 8
#define K 4

int main() {
    double *A_h, *B_h, *C_h;
    checkCudaErrors(hipHostMalloc((void**)&A_h, M*K*sizeof(double)));
    checkCudaErrors(hipHostMalloc((void**)&B_h, K*N*sizeof(double)));
    checkCudaErrors(hipHostMalloc((void**)&C_h, M*N*sizeof(double)));

    for(int i=0; i<M*K; i++) {
        A_h[i] = i;
    }
    for(int i=0; i<K*N; i++) {
        B_h[i] = i;
    }
    for(int i=0; i<M*N; i++) {
        C_h[i] = 0;
    }

    double *A = NULL;
    double *B = NULL;
    double *C = NULL;
    double *D = NULL;
    checkCudaErrors(hipMalloc((void**)&A, M*K*sizeof(double)));
    checkCudaErrors(hipMalloc((void**)&B, K*N*sizeof(double)));
    checkCudaErrors(hipMalloc((void**)&C, M*N*sizeof(double)));
    checkCudaErrors(hipMalloc((void**)&D, M*N*sizeof(double)));
    assert(((unsigned long long)A)%128 == 0);
    assert(((unsigned long long)B)%128 == 0);
    assert(((unsigned long long)C)%128 == 0);
    assert(((unsigned long long)D)%128 == 0);

    checkCudaErrors(hipMemcpy(A, A_h, M*K*sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(B, B_h, K*N*sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(C, C_h, M*N*sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemset(D, 0, M*N*sizeof(double)));

    


}